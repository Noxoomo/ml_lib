#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#if __CUDA_ARCH__ < 350
template <typename T>
__forceinline__ __device__ T __ldg(const T* data) {
    return data[0];
}
#endif

__forceinline__ __device__ float Sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

__forceinline__ __device__ float SigmoidDer(float x) {
    const float p = 1.0f / (1.0f + expf(-x));
    return p * (1.0f - p);
}

__global__ void PolynomProbsImpl(
    const float* features,
    int batchSize,
    const int* splits,
    const float* conditions,
    const int* polynomOffsets,
    int polynomCount,
    float lambda,
    float* probs) {
    if (threadIdx.x < batchSize) {
        int polynomId = blockIdx.x;

        features +=  threadIdx.x;
        probs += threadIdx.x;

        while (polynomId < polynomCount) {
            int offset = polynomOffsets[polynomId];
            int nextOffset = polynomOffsets[polynomId + 1];
            const int depth = nextOffset - offset;

            float logProb = 0;
            for (int i = 0; i < depth; ++i) {
                const int f = __ldg(splits + offset + i);
                const float c = __ldg(conditions + offset + i);
                const float x = __ldg(features + f * batchSize);
                const float val = -lambda * (x - c);
                const float expVal = 1.0f + exp(val);

//            p( split = 1) = 1.0 / (1.0 + exp(-(x - c)))
//            c = 0, x= inf, p = 1.0 / (1.0 + exp(-inf) = 0
//            log(p) = -log(1.0 + exp(-(x - c))
                const float isTrueLogProb = isfinite(expVal) ? log(expVal) : val;
                logProb -= isTrueLogProb;
            }
            const float prob = exp(logProb);
            probs[polynomId * batchSize] = prob;
            polynomId += gridDim.x;
        }
    }
}

//batch size should be equal to BlockSize
//we need to reduce polynoms for each output dim
__global__ void PolynomForwardImpl(
    const float* probs,
    int batchSize,
    const float* values,
    int polynomCount,
    int outputDim,
    float* out) {

    //out: batch_elem0 dim0, dim1, dimk batch_elem1 dim0 dim1 dimk
    //so threads
    int polynomId = blockIdx.x;
    const int dimId = blockIdx.y;

    int tid = threadIdx.x;
    if (tid >= batchSize) {
        return;
    }

    float sum = 0;
    probs += threadIdx.x;
    values += dimId;

    while (polynomId < polynomCount) {
        const float polynomProb = __ldg(probs + polynomId * batchSize);
        const float out = __ldg(values + polynomId * outputDim);
        sum += polynomProb * out;
        polynomId += gridDim.x;
    }

    atomicAdd(out + dimId * batchSize + threadIdx.x, sum);
}


//

//
void PolynomForward(
    const float lambda,
    const float* features,
    int fCount,
    int batchSize,
    const int* splits,
    const float* conditions,
    const int* polynomOffsets,
    const float* values,
    int polynomCount,
    int outDim,
    float* tempProbs,
    float* output
) {
    const int blockSize = batchSize;
    const int numBlocks = min(polynomCount, 1000);
    assert(batchSize < 2048);
    assert(numBlocks);

    PolynomProbsImpl << < numBlocks, blockSize >>> (features, batchSize, splits, conditions, polynomOffsets, polynomCount, lambda, tempProbs);

    dim3 forwardBlocks;
    forwardBlocks.z = 1;
    forwardBlocks.y = outDim;
    forwardBlocks.x = min(polynomCount, 512);
    PolynomForwardImpl << < forwardBlocks, batchSize >> > (tempProbs, batchSize, values, polynomCount, outDim, output);
}

//
//
///*
// * Here layout is not the same as in forward pass
// * BlockSize = 256, MaxDepth = 6, K = 24
// * should give 50% occupancy, this should be enough
// */
//template <int MaxDepth, int BlockSize, int K>
//__global__ void PolynomBackwardImpl(const float* features,
//                                    int featuresCount,
//                                    const float* outDer,
//                                    int outputDim,
//                                    const float* leafSum,
//                                    int* polynomDepths,
//                                    int* polynomOffset,
//                                    int* featureIds,
//                                    float* conditions,
//                                    int polynomCount,
//                                    float* out) {
//    const int sampleId = blockIdx.y;
//    features += sampleId * featuresCount;
//    out += sampleId * featuresCount;
//
//    outDer += sampleId * outputDim;
////    float outputDer = 0;
////    for (int dim = 0; dim < outputDim; ++dim) {
////        outputDer += outDer[dim];
////    }
//
//    //out: batch_elem0 dim0, dim1, dimk batch_elem1 dim0 dim1 dimk
//    //so threads
//
//    __shared__ float localFeaturesDer[BlockSize * K];
//    for (int i = threadIdx.x; i < BlockSize * K; i += BlockSize) {
//        localFeaturesDer[i] = 0;
//    }
//    __syncthreads();
//    const int alignedFeaturesCount = ((featuresCount + BlockSize - 1) / BlockSize) * BlockSize;
//    const int memoryBlocks = BlockSize * K / alignedFeaturesCount;
//    const int memoryBlockId = threadIdx.x % memoryBlocks;
//
//
//    int polynomId = blockIdx.x * gridDim.x + threadIdx.x;
//
//    while (polynomId < polynomCount) {
//        const int depth = polynomDepths[polynomId];
//        int offset = polynomOffset[polynomId];
//
//
//        float logOneMinusProb[MaxDepth];
//        short fids[MaxDepth];
//        float totalLogProb = 0;
//
//        #pragma unroll
//        for (int i = 0; i < MaxDepth; ++i) {
//            if (i < depth) {
//                const int f = __ldg(featureIds + i + offset);
//                fids[i] = f;
//                const float c = __ldg(conditions + i + offset);
//                const float x = __ldg(features + f);
//                const float val = -(x - c);
//                const float expVal = 1.0f + exp(val);
//                const float isTrueLogProb = (isfinite(expVal) ? log(expVal) : val);
//                totalLogProb += isTrueLogProb;
//                logOneMinusProb[i] = val - isTrueLogProb;
//            }
//        }
//
//        //featureDerivative is outputDer * total value before monom * monom derivative
//        float derMultiplier  = 0;
//        for (int dim = 0; dim < outputDim; ++dim) {
//            derMultiplier += __ldg(leafSum + polynomId * outputDim + dim) * __ldg(outDer + dim);
//        }
//
//        #pragma unroll
//        for (int i = 0; i < MaxDepth; ++i) {
//            if (i < depth) {
//                const int f = fids[i];
//                const int featureDer = exp(totalLogProb + logOneMinusProb[i]) * derMultiplier;
//                //atomics in shared memory, pretty fast on pascal+ hardware
//                atomicAdd(localFeaturesDer + memoryBlocks * f + memoryBlockId, featureDer);
//            }
//        }
//        polynomId += gridDim.x * blockDim.x;
//    }
//
//
//    __syncthreads();
//
//    //outputDim = 1024 => memoryBlocks = 6
//    for (int i = threadIdx.x; i < featuresCount; i += BlockSize) {
//        float der = 0;
//
//        for (int k = 0; k < memoryBlocks; ++k) {
//            der += localFeaturesDer[i * memoryBlocks + k];
//        }
//        atomicAdd(out + i,  localFeaturesDer[i * memoryBlocks + i]);
//    }
//}
//
//void PolynomBackward(const float* features,
//                     int featuresCount,
//                     int batchSize,
//                     const float* outDer,
//                     int outputDim,
//                     const float* leafSum,
//                     int* polynomDepths,
//                     int* polynomOffset,
//                     int* featureIds,
//                     float* conditions,
//                     int polynomCount,
//                     float* out,
//                     hipStream_t stream) {
//
//    const int blockSize = 256;
//    dim3 numBlocks;
//    numBlocks.z = 1;
//    numBlocks.y = batchSize;
//    //should be ≈ smCount * 6 / batchSize
//    numBlocks.x = (polynomCount + blockSize - 1) / blockSize;
//
//    const int maxDepth = 6;
//    const int K = 16;
//    PolynomBackwardImpl<maxDepth, blockSize, K> <<<numBlocks, blockSize, 0, stream >>>(features, featuresCount, outDer, outputDim,
//        leafSum, polynomDepths, polynomOffset, featureIds, conditions, polynomCount, out);
//
//}
//

